#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "ant.h"

double g_Trial[N_CITY_COUNT][N_CITY_COUNT]; //pheromone between every 2 cities
double g_Distance[N_CITY_COUNT][N_CITY_COUNT]; //distance between every 2 cities

//tsp城市坐标数据
double x_Ary[N_CITY_COUNT],y_Ary[N_CITY_COUNT];

//返回浮点数四舍五入取整后的浮点数
double ROUND(double dbA)
{
    return (double)((int)(dbA+0.5));
}

class CTsp
{
public:
    CTsp(void);
    ~CTsp(void);

public:
    CAnt m_cAntAry[N_ANT_COUNT]; //蚂蚁数组
    CAnt m_cBestAnt; //定义一个蚂蚁变量，用来保存搜索过程中的最优结果
                                        //该蚂蚁不参与搜索，只是用来保存最优结果

public:

    //city position
    void readTsp();

    //初始化数据
    void InitData();

    //开始搜索
    void Search();

    //更新环境信息素
    void UpdateTrial();

};

//构造函数
CTsp::CTsp(void)
{
}

CTsp::~CTsp(void)
{
}

void CTsp::readTsp()
{
	int i,j;

	FILE *fp=fopen("tsp/lin318.tsp","r") ;
	if(fp == NULL)
	{
		printf("sorry,file not found!\n");
		exit(0);
	}

	for(i=0;i<N_CITY_COUNT;i++)
	{
		fscanf(fp,"%d%lf%lf",&j,&x_Ary[i],&y_Ary[i]);
	}

	fclose(fp);
}

//初始化数据
void CTsp::InitData()
{
	//read tsp file
	readTsp();

    //先把最优蚂蚁的路径长度设置成一个很大的值
    m_cBestAnt.m_dbPathLength=DB_MAX;

    //计算两两城市间距离
    double dbTemp=0.0;
    for (int i=0;i<N_CITY_COUNT;i++)
    {
        for (int j=0;j<N_CITY_COUNT;j++)
        {
            dbTemp=(x_Ary[i]-x_Ary[j])*(x_Ary[i]-x_Ary[j])+(y_Ary[i]-y_Ary[j])*(y_Ary[i]-y_Ary[j]);
            dbTemp=pow(dbTemp,0.5);
            g_Distance[i][j]=ROUND(dbTemp);
        }
    }

    //初始化环境信息素，先把城市间的信息素设置成一样
    //这里设置成1.0，设置成多少对结果影响不是太大，对算法收敛速度有些影响
    for (int i=0;i<N_CITY_COUNT;i++)
    {
        for (int j=0;j<N_CITY_COUNT;j++)
        {
            g_Trial[i][j]=1.0;
        }
    }

}

//更新环境信息素
/*void CTsp::UpdateTrial()
{
    //临时数组，保存各只蚂蚁在两两城市间新留下的信息素
    double dbTempAry[N_CITY_COUNT][N_CITY_COUNT];
    memset(dbTempAry,0,sizeof(dbTempAry)); //先全部设置为0

    //计算新增加的信息素,保存到临时数组里
    int m=0;
    int n=0;
    for (int i=0;i<N_ANT_COUNT;i++) //计算每只蚂蚁留下的信息素
    {
            for (int j=1;j<N_CITY_COUNT;j++)
            {
                m=m_cAntAry[i].m_nPath[j];
                n=m_cAntAry[i].m_nPath[j-1];
                dbTempAry[n][m]=dbTempAry[n][m]+DBQ/m_cAntAry[i].m_dbPathLength;
                dbTempAry[m][n]=dbTempAry[n][m];
            }

            //最后城市和开始城市之间的信息素
            n=m_cAntAry[i].m_nPath[0];
            dbTempAry[n][m]=dbTempAry[n][m]+DBQ/m_cAntAry[i].m_dbPathLength;
            dbTempAry[m][n]=dbTempAry[n][m];

    }

    //==================================================================
    //更新环境信息素
    for (int i=0;i<N_CITY_COUNT;i++)
    {
        for (int j=0;j<N_CITY_COUNT;j++)
        {
            g_Trial[i][j]=g_Trial[i][j]*ROU+dbTempAry[i][j]; //最新的环境信息素 = 留存的信息素 + 新留下的信息素
        }
    }

}*/

void CTsp::UpdateTrial()
{
    double dbTempAry[N_CITY_COUNT][N_CITY_COUNT];
    memset(dbTempAry, 0 , sizeof(dbTempAry));

    int m = 0;
    int n = 0;
    for(int i=1; i<N_CITY_COUNT; i++)
    {
        m = m_cBestAnt.m_nPath[i];
        n = m_cBestAnt.m_nPath[i-1];
        dbTempAry[n][m] += DBQ/m_cBestAnt.m_dbPathLength;
        dbTempAry[m][n] = dbTempAry[n][m];
    }
    n = m_cBestAnt.m_nPath[0];
    dbTempAry[n][m] += DBQ/m_cBestAnt.m_dbPathLength;
    dbTempAry[m][n] = dbTempAry[n][m];

    for(int i=0; i<N_CITY_COUNT; i++)
    {
        for(int j=0; j<N_CITY_COUNT; j++)
        {
            g_Trial[i][j] = g_Trial[i][j]*ROU + dbTempAry[i][j];
        }
    }
}

void CTsp::Search()
{

    char cBuf[256]; //打印信息用

    //在迭代次数内进行循环
    for (int i=0;i<N_IT_COUNT;i++)
    {
        //每只蚂蚁搜索一遍
        for (int j=0;j<N_ANT_COUNT;j++)
        {
            m_cAntAry[j].Search(g_Trial, g_Distance);
        }

        //保存最佳结果
        for (int j=0;j<N_ANT_COUNT;j++)
        {
            if (m_cAntAry[j].m_dbPathLength < m_cBestAnt.m_dbPathLength)
            {
                m_cBestAnt=m_cAntAry[j];
            }
        }

        //更新环境信息素
        UpdateTrial();

        //输出目前为止找到的最优路径的长度
        sprintf(cBuf,"\n[%d] %.0f",i+1,m_cBestAnt.m_dbPathLength);
        printf(cBuf);
    }

}

int main()
{
	printf("Ants start searching tours . . .");

	//count running time
	clock_t MyTime, UTime;
	double duration = 0.0;
	MyTime = clock();

    //用当前时间点初始化随机种子，防止每次运行的结果都相同
    time_t tm;
    time(&tm);
    unsigned int nSeed=(unsigned int)tm;
    srand(nSeed);

    //开始搜索
    CTsp tsp;

    tsp.InitData(); //初始化
    tsp.Search(); //开始搜索

    //输出结果
    /*printf("\nThe best tour is :\n");

    char cBuf[128];
    for (int i=0;i<N_CITY_COUNT;i++)
    {
        sprintf(cBuf,"d ",tsp.m_cBestAnt.m_nPath[i]+1);
        if (i % 20 == 0)
        {
            printf("\n");
        }
        printf(cBuf);
    }*/
    UTime = clock();
    duration = (double)(UTime - MyTime) / CLOCKS_PER_SEC;
    printf("\nTotal time is %0.3f seconds\n", duration);
    printf("\nAnts' searching is done!\n");

    return 0;
}
