
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand.h> //for parallel cuRand

//const double ALPHA=2.0; //启发因子，信息素的重要程度
//const double BETA=3.0;   //期望因子，城市间距离的重要程度
const double ROU=0.5; //信息素残留参数

const int N_ANT_COUNT=256; //蚂蚁数量
const int N_IT_COUNT=15; //迭代次数
const int N_CITY_COUNT=150; //城市数量

const double DBQ=100.0; //总的信息素
const double DB_MAX=10e9; //一个标志数，10的9次方

double g_Trial[N_CITY_COUNT][N_CITY_COUNT]; //两两城市间信息素，就是环境信息素
double g_Distance[N_CITY_COUNT][N_CITY_COUNT]; //两两城市间距离

//for parallel cuRand
float *devData;

//data on device
double *d_Distance,*d_Trial;

//tsp城市坐标数据
double x_Ary[N_CITY_COUNT],y_Ary[N_CITY_COUNT];

//返回指定范围内的随机整数
int rnd(int nLow,int nUpper)
{
    return (int)(nLow+(nUpper-nLow)*rand()/((double)RAND_MAX+1.0));
}

//返回0~1范围内的随机浮点数(device)
void dev_rnd(unsigned int nSeed)
{
	hiprandGenerator_t gen;

	//hostData = (float *)calloc(N_ANT_COUNT*N_CITY_COUNT, sizeof(float));
	hipMalloc((void **)&devData,N_ANT_COUNT*N_CITY_COUNT*sizeof(float));

	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,nSeed);
	hiprandGenerateUniform(gen,devData,N_ANT_COUNT*N_CITY_COUNT);
//	cudaMemcpy(hostData,devData,N_ANT_COUNT*N_CITY_COUNT * sizeof(float),
//	        cudaMemcpyDeviceToHost);

	hiprandDestroyGenerator(gen);
}

//返回浮点数四舍五入取整后的浮点数
double ROUND(double dbA)
{
    return (double)((int)(dbA+0.5));
}

//定义蚂蚁类
class CAnt
{
public:
    CAnt(void);
    ~CAnt(void);

public:

    int m_nPath[N_CITY_COUNT]; //蚂蚁走的路径
    double m_dbPathLength; //蚂蚁走过的路径长度

    int m_nAllowedCity[N_CITY_COUNT]; //没去过的城市
    int m_nCurCityNo; //当前所在城市编号
    int m_nMovedCityCount; //已经去过的城市数量

public:

    __device__ void antInit(int antID,float *devData);  //初始化
    __device__ void antMove(int antID,double *d_Distance,double *d_Trial,float *devData); //蚂蚁在城市间移动
    __device__ void antCalPathLength(int antID,double *d_Distance);  //计算蚂蚁走过的路径长度
    __device__ int antChooseNextCity(int antID,int count,double *d_Distance,double *d_Trial,float *devData);
};

//构造函数
CAnt::CAnt(void)
{
}

//析构函数
CAnt::~CAnt(void)
{
}

//////////////////////////////////////////////////////////
//device or kernel functions

//初始化函数，蚂蚁搜索前调用
__device__
void CAnt:: antInit(int antID,float *devData)
{
    for (int i=0;i<N_CITY_COUNT;i++)
    {
        m_nAllowedCity[i]=1; //设置全部城市为没有去过
        m_nPath[i]=0; //蚂蚁走的路径全部设置为0
    }

    //蚂蚁走过的路径长度设置为0
    m_dbPathLength=0.0;

    //随机选择一个出发城市
    m_nCurCityNo=(int)N_CITY_COUNT*devData[antID];

    //把出发城市保存入路径数组中
    m_nPath[0]=m_nCurCityNo;

    //标识出发城市为已经去过了
    m_nAllowedCity[m_nCurCityNo]=0;

    //已经去过的城市数量设置为1
    m_nMovedCityCount=1;
}

//选择下一个城市
//返回值 为城市编号
__device__
int CAnt::antChooseNextCity(int antID,int count,double *d_Distance,double *d_Trial,float *devData)
{
    int nSelectedCity=-1; //返回结果，先暂时把其设置为-1

    //==============================================================================
    //计算当前城市和没去过的城市之间的信息素总和

    double dbTotal=0.0,tA,tB;
    double prob[N_CITY_COUNT]; //保存各个城市被选中的概率

    for (int i=0;i<N_CITY_COUNT;i++)
    {
        if (m_nAllowedCity[i] == 1) //城市没去过
        {
            //prob[i]=pow(d_Trial[m_nCurCityNo*N_CITY_COUNT+i],ALPHA)*pow(1.0/d_Distance[m_nCurCityNo*N_CITY_COUNT+i],BETA); //该城市和当前城市间的信息素
        	tA=d_Trial[m_nCurCityNo*N_CITY_COUNT+i];
        	tB=1.0/d_Distance[m_nCurCityNo*N_CITY_COUNT+i];
        	prob[i]=(tA*tA)*(tB*tB*tB); //ALPHA=2.0,BETA=3.0
        	dbTotal=dbTotal+prob[i]; //累加信息素，得到总和
        }
        else //如果城市去过了，则其被选中的概率值为0
        {
            prob[i]=0.0;
        }
    }

    //==============================================================================
    //进行轮盘选择
    double dbTemp=0.0;
    if (dbTotal > 0.0) //总的信息素值大于0
    {
        //取一个随机数devData
    	dbTemp=dbTotal * devData[m_nMovedCityCount+antID*N_CITY_COUNT-1];

        for (int i=0;i<N_CITY_COUNT;i++)
        {
            if (m_nAllowedCity[i] == 1) //城市没去过
            {
                dbTemp=dbTemp-prob[i]; //这个操作相当于转动轮盘，如果对轮盘选择不熟悉，仔细考虑一下
                if (dbTemp < 0.0) //轮盘停止转动，记下城市编号，直接跳出循环
                {
                    nSelectedCity=i;
                    break;
                }
            }
        }
    }

    //==============================================================================
    //如果城市间的信息素非常小 ( 小到比double能够表示的最小的数字还要小 )
    //那么由于浮点运算的误差原因，上面计算的概率总和可能为0
    //会出现经过上述操作，没有城市被选择出来
    //出现这种情况，就把第一个没去过的城市作为返回结果

    //题外话：刚开始看的时候，下面这段代码困惑了我很长时间，想不通为何要有这段代码，后来才搞清楚。
    if (nSelectedCity == -1)
    {
        for (int i=0;i<N_CITY_COUNT;i++)
        {
            if (m_nAllowedCity[i] == 1) //城市没去过
            {
                nSelectedCity=i;
                break;
            }
        }
    }

    //==============================================================================
    //返回结果，就是城市的编号
    return nSelectedCity;
}

//蚂蚁在城市间移动
__device__
void CAnt::antMove(int antID,double *d_Distance,double *d_Trial,float *devData)
{
    int nCityNo=antChooseNextCity(antID,m_nMovedCityCount,d_Distance,d_Trial,devData); //选择下一个城市

    m_nPath[m_nMovedCityCount]=nCityNo; //保存蚂蚁走的路径
    m_nAllowedCity[nCityNo]=0; //把这个城市设置成已经去过了
    m_nCurCityNo=nCityNo; //改变当前所在城市为选择的城市
    m_nMovedCityCount++; //已经去过的城市数量加1
}

//计算蚂蚁走过的路径长度
__device__
void CAnt::antCalPathLength(int antID,double *d_Distance)
{
    m_dbPathLength=0.0; //先把路径长度置0
    int m=0;
    int n=0;

    for (int i=1;i<N_CITY_COUNT;i++)
    {
        m=m_nPath[i];
        n=m_nPath[i-1];
        m_dbPathLength=m_dbPathLength+d_Distance[m*N_CITY_COUNT+n];
    }

    //加上从最后城市返回出发城市的距离
    n=m_nPath[0];
    m_dbPathLength=m_dbPathLength+d_Distance[m*N_CITY_COUNT+n];
}

//蚂蚁进行搜索
__global__
void antSearch_Kernel(CAnt *d_AntAry,double *d_Distance,double *d_Trial,float *devData)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;

	if(i < N_ANT_COUNT)
	{
		d_AntAry[i].antInit(i,devData); //initialize data for every ant

		while(d_AntAry[i].m_nMovedCityCount < N_CITY_COUNT)
		{
			d_AntAry[i].antMove(i,d_Distance,d_Trial,devData);
		}
		d_AntAry[i].antCalPathLength(i,d_Distance);
	}
}

///////////////////////////////////////////////////////////////////////////////

//tsp类
class CTsp
{
public:
    CTsp(void);
    ~CTsp(void);

public:
    CAnt m_cAntAry[N_ANT_COUNT]; //蚂蚁数组(host)
    CAnt m_cBestAnt; //定义一个蚂蚁变量，用来保存搜索过程中的最优结果
                                        //该蚂蚁不参与搜索，只是用来保存最优结果

public:

    //get city position
    void readTsp();

    //初始化数据
    void InitData();

    //开始搜索
    void Search();

    //更新环境信息素
    void UpdateTrial();

    //搜索路径
    void antSearch(); //kernel inside

};

//构造函数
CTsp::CTsp(void)
{
}

CTsp::~CTsp(void)
{
}

//read tsp file
void CTsp::readTsp()
{
	int i,j;

	FILE *fp=fopen("tsp/ch150.tsp","r") ;
	if(fp == NULL)
	{
		printf("sorry,file not found!\n");
		exit(0);
	}

	for(i=0;i<N_CITY_COUNT;i++)
	{
		fscanf(fp,"%d%lf%lf",&j,&x_Ary[i],&y_Ary[i]);
	}

	fclose(fp);
}

CAnt *d_AntAry; //ants on GPU

//初始化数据
void CTsp::InitData()
{
	//read tsp file
	readTsp();

    //先把最优蚂蚁的路径长度设置成一个很大的值
    m_cBestAnt.m_dbPathLength=DB_MAX;

    //计算两两城市间距离
    double dbTemp=0.0;
    for (int i=0;i<N_CITY_COUNT;i++)
    {
        for (int j=0;j<N_CITY_COUNT;j++)
        {
        	//compute distance between cities
            dbTemp=(x_Ary[i]-x_Ary[j])*(x_Ary[i]-x_Ary[j])+(y_Ary[i]-y_Ary[j])*(y_Ary[i]-y_Ary[j]);
            dbTemp=sqrt(dbTemp);
            g_Distance[i][j]=ROUND(dbTemp);
        }
    }

    //初始化环境信息素，先把城市间的信息素设置成一样
    //这里设置成1.0，设置成多少对结果影响不是太大，对算法收敛速度有些影响
    for (int i=0;i<N_CITY_COUNT;i++)
    {
        for (int j=0;j<N_CITY_COUNT;j++)
        {
            g_Trial[i][j]=1.0;
        }
    }
}

// every ant search as a thread
void CTsp::antSearch()
{
    //initialize data on GPU
    size_t size=sizeof(CAnt)*N_ANT_COUNT;
    hipMalloc(&d_AntAry,size);

    size=sizeof(double)*N_CITY_COUNT*N_CITY_COUNT;
    hipMalloc(&d_Distance,size);
    hipMalloc(&d_Trial,size);

	// data copy
	hipMemcpy(d_Distance,&g_Distance[0][0],size,hipMemcpyHostToDevice);
	hipMemcpy(d_Trial,&g_Trial[0][0],size,hipMemcpyHostToDevice);

	//kernel use
	antSearch_Kernel<<<ceil(N_ANT_COUNT/256.0), 256.0>>>(d_AntAry,d_Distance,d_Trial,devData);

	size=sizeof(CAnt)*N_ANT_COUNT;
	hipMemcpy(m_cAntAry, &d_AntAry[0],size,hipMemcpyDeviceToHost);
}

//更新环境信息素
void CTsp::UpdateTrial()
{
    //临时数组，保存各只蚂蚁在两两城市间新留下的信息素
    double dbTempAry[N_CITY_COUNT][N_CITY_COUNT];
    memset(dbTempAry,0,sizeof(dbTempAry)); //先全部设置为0

    //计算新增加的信息素,保存到临时数组里
    int m=0;
    int n=0;
    for (int i=0;i<N_ANT_COUNT;i++) //计算每只蚂蚁留下的信息素
    {
            for (int j=1;j<N_CITY_COUNT;j++)
            {
                m=m_cAntAry[i].m_nPath[j];
                n=m_cAntAry[i].m_nPath[j-1];
                dbTempAry[n][m]=dbTempAry[n][m]+DBQ/m_cAntAry[i].m_dbPathLength;
                dbTempAry[m][n]=dbTempAry[n][m];
            }

            //最后城市和开始城市之间的信息素
            n=m_cAntAry[i].m_nPath[0];
            dbTempAry[n][m]=dbTempAry[n][m]+DBQ/m_cAntAry[i].m_dbPathLength;
            dbTempAry[m][n]=dbTempAry[n][m];

    }

    //==================================================================
    //更新环境信息素
    for (int i=0;i<N_CITY_COUNT;i++)
    {
        for (int j=0;j<N_CITY_COUNT;j++)
        {
            g_Trial[i][j]=g_Trial[i][j]*ROU+dbTempAry[i][j]; //最新的环境信息素 = 留存的信息素 + 新留下的信息素
        }
    }
}

void CTsp::Search()
{

    char cBuf[256]; //打印信息用

    //在迭代次数内进行循环
    for (int i=0;i<N_IT_COUNT;i++)
    {
        //每只蚂蚁搜索一遍
        antSearch();

        //保存最佳结果
        for (int j=0;j<N_ANT_COUNT;j++)
        {
            if (m_cAntAry[j].m_dbPathLength < m_cBestAnt.m_dbPathLength)
            {
                m_cBestAnt=m_cAntAry[j];
            }
        }

        //更新环境信息素
        UpdateTrial();

        //输出目前为止找到的最优路径的长度
        sprintf(cBuf,"\n[%d] %.0f",i+1,m_cBestAnt.m_dbPathLength);
        printf(cBuf);
    }
}

int main()
{
    printf("Ants start searching tours . . .");

    //count running time
    clock_t MyTime, UTime;
    double duration = 0.0;
    MyTime = clock();

    //用当前时间点初始化随机种子，防止每次运行的结果都相同
    time_t tm;
    time(&tm);
    unsigned int nSeed=(unsigned int)tm;

    //generate random number for next city
    dev_rnd(nSeed);

    //开始搜索
    CTsp tsp;

    tsp.InitData(); //初始化
    tsp.Search(); //开始搜索

    //输出结果
    /*printf("\nThe best tour is :\n");

    char cBuf[128];
    for (int i=0;i<N_CITY_COUNT;i++)
    {
        sprintf(cBuf,"d ",tsp.m_cBestAnt.m_nPath[i]+1);
        if (i % 20 == 0)
        {
            printf("\n");
        }
        printf(cBuf);
    }*/

    UTime = clock();
    duration = (double)(UTime - MyTime) / CLOCKS_PER_SEC;
    printf("\nTotal time is %0.3f seconds\n", duration);
    printf("\nAnts' searching is done!\n");

    //release memory on device
    hipFree(devData);
    hipFree(d_Distance);
    hipFree(d_Trial);
    hipFree(d_AntAry);

    return 0;
}
